#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include<omp.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

void fill_matrix(int *A, int fac, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      A[i*n+j] = i+j*fac;
    }
  }
}

void print_matrix(int *A, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      printf("mat[%d, %d] = %d\n", i, j, A[i*n+j]);
    }
  }
}

//CPU version of the calculations
// just the product c_ij = Aij*B_ij
void perform_operation(int *A, int *B, int *C, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      C[i*n+j] = A[i*n+j]*B[i*n+j];
      //printf("C[%d, %d] = %d\n", i, j, C[i*n+j]);
    }
  }
}

//gpu version of the calculations
__global__ void perform_operation_cuda(int *A, int *B, int *C, int m, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i<m)
  {
    if (j<n)
    {
      C[i*n+j] = A[i*n+j]*B[i*n+j];
      //printf("C[%d, %d] = %d\n", i, j, C[i*n+j]);
    }
  }
}


//do the sum of the different C matrix in order to check the results
int do_sum(int *C, int sum, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      sum = sum + C[i*n+j];
    }
  }

  return sum;
}


//main test program
int main (void)
{
  int N = 3;
  int A[N*N], B[N*N], C[N*N];
  int f, nf=3, sum = 0, sum_ref = 0;
  int *A_d, *B_d, *C_d;

  dim3 dimBlock(N*N, N*N);
  dim3 dimGrid(1, 1);

  int num_gpus = 0;
  int gpuid = -1;
  unsigned int cpu_thread_id = -1;

  //initialisation matrices
  fill_matrix(A, 2, N, N);
  fill_matrix(B, 1, N, N);
  fill_matrix(C, 0, N, N);

  printf("Print A:\n");
  print_matrix(A, N, N);

  //run for checking
  for (f=0; f<nf; f++)
  {
    fill_matrix(B, f+1, N, N);
    perform_operation(A, B, C, N, N);
    sum_ref = do_sum(C, sum_ref, N, N);
  }
  printf("SUM_REF = %d\n", sum_ref);

  //end references


  //Set the threads to each GPUs
#pragma omp parallel private(num_gpus, cpu_thread_id, gpuid)
{
  hipGetDeviceCount(&num_gpus);
  cpu_thread_id = omp_get_thread_num();
  checkCudaErrors(hipSetDevice(cpu_thread_id % num_gpus));
  checkCudaErrors(hipGetDevice(&gpuid));
  printf("CPU thread %d uses CUDA device %d\n", cpu_thread_id, gpuid);
}

  //Start calculation with gpu
  sum = 0;

  checkCudaErrors(hipMalloc( (void **)&A_d, sizeof(int) * N*N)); //I want it here!!
  checkCudaErrors(hipMemcpy( A_d, A, sizeof(int) * N*N, hipMemcpyHostToDevice)); //I want it here!!

// We want A_d in shared not in private!!
#pragma omp parallel \
  shared(dimGrid, dimBlock, A_d, N, nf, sum) private(f, B, B_d, C_d, C)
{
  checkCudaErrors(hipMalloc( (void **)&B_d, sizeof(int) * N*N));
  checkCudaErrors(hipMalloc( (void **)&C_d, sizeof(int) * N*N));

  #pragma omp for reduction(+:sum)
  for (f=0; f<nf; f++)
  {
    fill_matrix(B, f+1, N, N);
    checkCudaErrors(hipMemcpy( B_d, B, sizeof(int) * N*N, hipMemcpyHostToDevice));
    //perform_operation(A, B, C, N, N);

    perform_operation_cuda<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N, N);
    checkCudaErrors(hipMemcpy( C, C_d, sizeof(int) * N*N, hipMemcpyDeviceToHost));
    sum = do_sum(C, sum, N, N);
  }
  checkCudaErrors(hipFree(B_d));
  checkCudaErrors(hipFree(C_d));
}
  checkCudaErrors(hipFree(A_d));

  //check
  printf("SUM = %d\n", sum);
  printf("SUM - SUM_REF = %d\n", sum-sum_ref);
  checkCudaErrors(hipDeviceReset());

  return 0;
}
